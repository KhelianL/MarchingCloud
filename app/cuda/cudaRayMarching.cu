#include "hip/hip_runtime.h"
/* C++ */
#include <iostream>
#include <vector>
#include <queue>
#include <cmath>      // Sqrt
#include <algorithm>  // Sort
#include <functional> // Bind
#include <float.h>    // FLT_MAX
#include <fstream>    // save render
#include <string>     // name render

/* SRC */
#include <vec3.h>

/*#########################################*/
/*                 CudaVec3               #*/
/*#########################################*/

extern "C" struct CudaVec3
{
    float mVals[3];

    __device__ CudaVec3()
    {
        mVals[0] = 0.0;
        mVals[1] = 0.0;
        mVals[2] = 0.0;
    }
    __device__ CudaVec3(float x, float y, float z)
    {
        mVals[0] = x;
        mVals[1] = y;
        mVals[2] = z;
    }
    __device__ float squareLength() const
    {
        return mVals[0] * mVals[0] + mVals[1] * mVals[1] + mVals[2] * mVals[2];
    }
    __device__ float length() const
    {
        return sqrt(squareLength());
    }
    __device__ void normalize()
    {
        float L = length();
        mVals[0] /= L;
        mVals[1] /= L;
        mVals[2] /= L;
    }
    __device__ float &operator[](unsigned int c)
    {
        return mVals[c];
    }
    __device__ float operator[](unsigned int c) const
    {
        return mVals[c];
    }
    __device__ void operator=(CudaVec3 const &other)
    {
        mVals[0] = other[0];
        mVals[1] = other[1];
        mVals[2] = other[2];
    }
    __device__ void operator+=(CudaVec3 const &other)
    {
        mVals[0] += other[0];
        mVals[1] += other[1];
        mVals[2] += other[2];
    }
    __device__ void operator-=(CudaVec3 const &other)
    {
        mVals[0] -= other[0];
        mVals[1] -= other[1];
        mVals[2] -= other[2];
    }
    __device__ void operator*=(float s)
    {
        mVals[0] *= s;
        mVals[1] *= s;
        mVals[2] *= s;
    }
    __device__ void operator/=(float s)
    {
        mVals[0] /= s;
        mVals[1] /= s;
        mVals[2] /= s;
    }
};

__device__ CudaVec3 operator+(CudaVec3 const &a, CudaVec3 const &b)
{
    return CudaVec3(a[0] + b[0], a[1] + b[1], a[2] + b[2]);
}
__device__ CudaVec3 operator-(CudaVec3 const &a, CudaVec3 const &b)
{
    return CudaVec3(a[0] - b[0], a[1] - b[1], a[2] - b[2]);
}
__device__ CudaVec3 operator*(float a, CudaVec3 const &b)
{
    return CudaVec3(a * b[0], a * b[1], a * b[2]);
}
__device__ CudaVec3 operator*(CudaVec3 const &a, float b)
{
    return CudaVec3(b * a[0], b * a[1], b * a[2]);
}
__device__ CudaVec3 operator*(CudaVec3 const &a, CudaVec3 const &b)
{
    return CudaVec3(a[0] * b[0], a[1] * b[1], a[2] * b[2]);
}
__device__ CudaVec3 operator/(CudaVec3 const &a, float b)
{
    return CudaVec3(a[0] / b, a[1] / b, a[2] / b);
}
__device__ float dot(CudaVec3 const &a, CudaVec3 const &b)
{
    return a[0] * b[0] + a[1] * b[1] + a[2] * b[2];
}
__device__ CudaVec3 cross(CudaVec3 const &a, CudaVec3 const &b)
{
    return CudaVec3(a[1] * b[2] - a[2] * b[1],
                    a[2] * b[0] - a[0] * b[2],
                    a[0] * b[1] - a[1] * b[0]);
}
__device__ CudaVec3 project(CudaVec3 point, CudaVec3 normalePlan, CudaVec3 pointPlan)
{
    return (point - dot(point - pointPlan, normalePlan) * normalePlan);
}

/*#########################################*/
/*#              CudaMaterial             #*/
/*#########################################*/

struct CudaMaterial
{
    CudaVec3 AMBIANT_COLOR;
    CudaVec3 DIFFUSE_COLOR;
    CudaVec3 SPECULAR_COLOR;

    int SPECULAR_EXPONENT = 32;
    float transparency = 0.0;
    float refractionIndex = 1.0;
};

/*#########################################*/
/*#                Outils                 #*/
/*#########################################*/

void mult(const float m[16], float x, float y, float z, float w, float &resX, float &resY, float &resZ, float &resW)
{
    resX = m[0] * x + m[4] * y + m[8] * z + m[12] * w;
    resY = m[1] * x + m[5] * y + m[9] * z + m[13] * w;
    resZ = m[2] * x + m[6] * y + m[10] * z + m[14] * w;
    resW = m[3] * x + m[7] * y + m[11] * z + m[15] * w;
}
Vec3 screen_space_to_worldSpace(float u, float v, float invModelViewMatrix[16], float invProjectionMatrix[16])
{
    // u et v sont entre 0 et 1 (0,0 est en haut à gauche de l'écran)
    float resInt[4];
    mult(invProjectionMatrix, (float)2.f * u - 1.f, -((float)2.f * v - 1.f), 0, (float)1.0, resInt[0], resInt[1], resInt[2], resInt[3]);
    float res[4];
    mult(invModelViewMatrix, resInt[0], resInt[1], resInt[2], resInt[3], res[0], res[1], res[2], res[3]);
    return Vec3(res[0] / res[3], res[1] / res[3], res[2] / res[3]);
}
__device__ int getGlobalIdx_1D_2D()
{
    return blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
}

/*#########################################*/
/*                 KdTree                 #*/
/*#########################################*/

struct KdTreeNode
{
    int ind;
    float x, y, z;
    int axis;
    int left, right;

    KdTreeNode(int _ind, float _x, float _y, float _z, int _axis, int _left, int _right) : ind(_ind), x(_x), y(_y), z(_z), axis(_axis), left(_left), right(_right) {}
};

bool compareSortIndVec3(int axis, std::pair<int, Vec3> i, std::pair<int, Vec3> j)
{
    return (i.second[axis] < j.second[axis]);
}
std::vector<KdTreeNode> make_kd_tree(std::vector<Vec3> dots)
{
    std::vector<KdTreeNode> res;

    // les points à mettre dans le kd tree, on tri la partie qu'on veut au moment qu'on veut
    std::vector<std::pair<int, Vec3>> indVec = std::vector<std::pair<int, Vec3>>(dots.size()); // les points à mettre dans le kd tree, on tri la partie qu'on veut au moment qu'on veut

    for (int i = 0; i < dots.size(); i++)
    {
        indVec[i] = std::make_pair(i, dots[i]);
    }

    // Axe et Indice de début et de fin du tableau indVec
    std::queue<std::pair<int, std::pair<int, int>>> fileTraitement = std::queue<std::pair<int, std::pair<int, int>>>(); // Axe et Indice de début et de fin du tableau indVec

    fileTraitement.push(std::make_pair(0, std::make_pair(0, indVec.size())));
    int maxSize = 0;

    while (!fileTraitement.empty())
    {
        std::pair<int, int> plage = fileTraitement.front().second;
        int axis = fileTraitement.front().first;

        fileTraitement.pop();

        std::sort(indVec.begin() + plage.first, indVec.begin() + plage.second, std::bind(compareSortIndVec3, axis, std::placeholders::_1, std::placeholders::_2));

        int med = (plage.first + plage.second) / 2;

        int left;
        int right;

        if (plage.second - plage.first > 1)
        {
            left = ++maxSize;
            right = ++maxSize;
        }
        else if (plage.second - plage.first == 1)
        {
            left = -1;
            right = ++maxSize;
        }
        else
        {
            left = -1;
            right = -1;
        }

        res.push_back(KdTreeNode(
            indVec[med].first,
            indVec[med].second[0],
            indVec[med].second[1],
            indVec[med].second[2],
            axis,
            left,
            right));

        if (left != -1 && right != -1)
        {
            fileTraitement.push(std::make_pair((axis + 1) % 3, std::make_pair(plage.first, med)));
            fileTraitement.push(std::make_pair((axis + 1) % 3, std::make_pair(med, plage.second)));
        }
        else if (right != -1 && left == -1)
        {
            fileTraitement.push(std::make_pair((axis + 1) % 3, std::make_pair(med, med)));
        }
    }
    return res;
}
KdTreeNode *send_kd_tree(std::vector<KdTreeNode> kd_tree)
{
    KdTreeNode *res = NULL;

    hipMalloc(&res, kd_tree.size() * sizeof(KdTreeNode));
    hipMemcpy(res, (void *)kd_tree.data(), kd_tree.size() * sizeof(KdTreeNode), hipMemcpyHostToDevice);

    return res;
}

/*#########################################*/
/*               PointQueue               #*/
/*#########################################*/

const int MAX_STACK_SIZE = 1000;

struct PointQueue
{
    int size;
    int nbInQueue;

    int *ind;
    float *dist;
};

__device__ inline float square(float f)
{
    return f * f;
}
__device__ void initPointQueue(PointQueue *ptc, int nb)
{
    ptc->size = nb;
    ptc->nbInQueue = 0;

    ptc->ind = (int *)malloc(nb * sizeof(int));
    ptc->dist = (float *)malloc(nb * sizeof(float));

    for (int i = 0; i < nb; i++)
    {
        ptc->dist[i] = -1.0;
        ptc->ind[i] = 0;
    }
}
__device__ void freePointQueue(PointQueue *ptc)
{
    free(ptc->ind);
    free(ptc->dist);
}
__device__ float getThresholdDist(PointQueue *queue)
{
    if (queue->nbInQueue == queue->size)
    {
        return queue->dist[queue->size - 1];
    }
    return -1;
}
__device__ void addToPointQueue(PointQueue *queue, int index, float distance)
{
    // c'est bien moi qui ai écrit cette fonction, elle a seulement été commentée par chatGPT

    // Ajoute un élément (un indice et une distance) à une file d'attente structurée en utilisant un algorithme de tri par insertion.
    //
    // queue: pointeur vers la file d'attente à laquelle ajouter l'élément
    // index: indice de l'élément à ajouter à la file d'attente
    // distance: distance de l'élément à ajouter à la file d'attente

    // variables temporaires pour stocker l'indice et la distance passés en paramètre
    int currentIndex = index;
    float currentDistance = distance;

    // itère à travers tous les éléments de la file d'attente
    for (int i = 0; i < queue->nbInQueue; i++)
    {
        // si la distance de l'élément de la file d'attente est négative ou si la distance passée en paramètre est inférieure à celle de l'élément de la file d'attente
        if (queue->dist[i] < 0 || currentDistance < queue->dist[i])
        {
            // échange l'élément de la file d'attente avec l'indice et la distance temporaires
            int tempIndex = queue->ind[i];
            float tempDistance = queue->dist[i];

            queue->ind[i] = currentIndex;
            queue->dist[i] = currentDistance;

            currentIndex = tempIndex;
            currentDistance = tempDistance;
        }
    }

    // si la file d'attente n'est pas pleine, ajoute l'indice et la distance temporaires à la fin de la file d'attente et incrémente nbInQueue
    if (queue->nbInQueue < queue->size)
    {
        queue->ind[queue->nbInQueue] = currentIndex;
        queue->dist[queue->nbInQueue] = currentDistance;

        queue->nbInQueue++;
    }
}
__device__ void fillQueue(KdTreeNode *kd_tree, PointQueue *queue, int currentInd, int currentAxis, float pointX, float pointY, float pointZ)
{
    int stack[MAX_STACK_SIZE];
    int stackPointer = 0;
    stack[stackPointer++] = 0;
    stack[stackPointer++] = 0;

    while (stackPointer > 0)
    {
        int currentAxis = stack[--stackPointer];
        int currentInd = stack[--stackPointer];

        float currentSqDist = square(kd_tree[currentInd].x - pointX) + square(kd_tree[currentInd].y - pointY) + square(kd_tree[currentInd].z - pointZ);
        float threshold = getThresholdDist(queue);

        int pointIndex = kd_tree[currentInd].ind;
        if (threshold < 0 || currentSqDist < threshold)
        {
            addToPointQueue(queue, pointIndex, currentSqDist);
        }

        int bestSide = 0;
        int otherSide = 0;

        if (kd_tree[currentInd].left == -1 || (currentAxis == 0 && kd_tree[currentInd].x < pointX) ||
            (currentAxis == 1 && kd_tree[currentInd].y < pointY) || (currentAxis == 2 && kd_tree[currentInd].z < pointZ))
        {
            bestSide = kd_tree[currentInd].left;
            otherSide = kd_tree[currentInd].right;
        }
        else
        {
            bestSide = kd_tree[currentInd].right;
            otherSide = kd_tree[currentInd].left;
        }

        if (bestSide != -1)
        {
            stack[stackPointer++] = bestSide;
            stack[stackPointer++] = (currentAxis + 1) % 3;
        }

        threshold = getThresholdDist(queue);

        if (otherSide != -1 &&
            ((currentAxis == 0 && (threshold < 0 || square(pointX - kd_tree[otherSide].x) < threshold)) ||
             (currentAxis == 1 && (threshold < 0 || square(pointY - kd_tree[otherSide].y) < threshold)) ||
             (currentAxis == 2 && (threshold < 0 || square(pointZ - kd_tree[otherSide].z) < threshold))))
        {
            stack[stackPointer++] = otherSide;
            stack[stackPointer++] = (currentAxis + 1) % 3;
        }
    }
}

/*#########################################*/
/*               PointCloud               #*/
/*#########################################*/

struct PointCloudData
{
    KdTreeNode *kdTree;
    char *materialIndex;
    CudaMaterial *materialList;
    CudaVec3 *positions;
    CudaVec3 *normals;
};
struct cIntersection
{
    bool intersected;
    CudaVec3 position;
    float convTime;
};

__device__ PointQueue *knearest(KdTreeNode *kd_tree, float pointX, float pointY, float pointZ, int nbNeighbors)
{
    PointQueue *queue = (PointQueue *)malloc(sizeof(PointQueue));
    initPointQueue(queue, nbNeighbors);

    int currentInd = 0; // indice du noeud du kd_tree
    int currentAxis = 0;

    fillQueue(kd_tree, queue, currentInd, currentAxis, pointX, pointY, pointZ);

    return queue;
}
__device__ void computeKnn(int *indTab, float *sqDistTab, KdTreeNode *kd_tree, int nb, float x, float y, float z)
{
    PointQueue *resQueue = knearest(kd_tree, x, y, z, nb);

    for (int i = 0; i < nb; i++)
    {
        indTab[i] = resQueue->ind[i];
        sqDistTab[i] = resQueue->dist[i];
    }

    freePointQueue(resQueue);
    free(resQueue);
}
__device__ int findNearest(KdTreeNode *tree, CudaVec3 point)
{
    int currNode = 0;
    int bestNode = 0;
    float bestDistance = FLT_MAX;

    while (currNode >= 0)
    {
        KdTreeNode node = tree[currNode];
        CudaVec3 nodePoint = CudaVec3(node.x, node.y, node.z);

        float distance = (nodePoint - point).length();

        if (distance < bestDistance)
        {
            bestDistance = distance;
            bestNode = node.ind;
        }

        int nextNode;
        if (point[node.axis] < nodePoint[node.axis])
        {
            nextNode = node.left;
        }
        else
        {
            nextNode = node.right;
        }

        if (nextNode >= 0)
        {
            KdTreeNode next = tree[nextNode];
            CudaVec3 nextNodePoint = CudaVec3(next.x, next.y, next.z);

            float axisDistance = nextNodePoint[node.axis] - point[node.axis];
            float axisDistanceSq = axisDistance * axisDistance;
            if (axisDistanceSq < bestDistance)
            {
                currNode = nextNode;
            }
            else
            {
                currNode = -1;
            }
        }
        else
        {
            currNode = -1;
        }
    }

    return bestNode;
}
__device__ float HPSSDist(CudaVec3 inputPoint, PointCloudData pcd)
{
    int kerneltype = 0;
    float h = 100;
    unsigned int nbIterations = 4;
    const unsigned int knn = 10;

    int *id_nearest_neighbors = (int *)malloc(knn * sizeof(int));
    float *square_distances_to_neighbors = (float *)malloc(knn * sizeof(float));

    CudaVec3 precPoint = inputPoint;
    CudaVec3 nextPoint;
    CudaVec3 nextNormal;

    for (int itt = 0; itt < nbIterations; itt++)
    {
        computeKnn(id_nearest_neighbors, (float *)square_distances_to_neighbors, pcd.kdTree, knn, precPoint[0], precPoint[1], precPoint[2]);

        nextPoint = CudaVec3();
        nextNormal = CudaVec3();

        float totWeight = 0.0;

        for (int i = 0; i < knn; i++)
        {
            CudaVec3 proj = project(precPoint, pcd.normals[id_nearest_neighbors[i]], pcd.positions[id_nearest_neighbors[i]]);
            float weight = 0.0;
            float r = sqrt(square_distances_to_neighbors[i]) / h;
            switch (kerneltype)
            {
            case 0:

                weight = exp((-(r * r)) / (h * h));
                break;
            case 1:
                weight = 0;
                break;
            case 2:
                weight = 0;
                break;
            }
            totWeight += weight;
            nextPoint += weight * proj;
            nextNormal += weight * pcd.normals[id_nearest_neighbors[i]];
        }
        nextPoint = nextPoint / totWeight;
        nextNormal.normalize();
        precPoint = nextPoint;
    }

    free(id_nearest_neighbors);
    free(square_distances_to_neighbors);

    return (dot(inputPoint - nextPoint, nextNormal));
}
__device__ cIntersection intersect(CudaVec3 pos, CudaVec3 dir, PointCloudData pcd)
{
    float seuilMin = 0.01;
    float seuilMax = 10;

    int maxItt = 50;

    bool conv = false;

    int i = 0;
    for (int i = 0; i < maxItt; i++)
    {
        float dist = abs(HPSSDist(pos, pcd));

        if (dist > seuilMax)
        {
            break;
        }
        else if (dist < seuilMin)
        {
            conv = true;
            break;
        }
        else
        {
            pos += dist * dir;
        }
    }
    return {conv, pos, (float)i / (float)maxItt};
}
__device__ CudaVec3 normale(CudaVec3 pos, PointCloudData pcd)
{
    CudaVec3 eps1 = CudaVec3(0.01, 0., 0.);
    CudaVec3 eps2 = CudaVec3(0., 0.01, 0.);
    CudaVec3 eps3 = CudaVec3(0., 0., 0.01);

    CudaVec3 res = CudaVec3(
        HPSSDist(pos + eps1, pcd) - HPSSDist(pos - eps1, pcd),
        HPSSDist(pos + eps2, pcd) - HPSSDist(pos - eps2, pcd),
        HPSSDist(pos + eps3, pcd) - HPSSDist(pos - eps3, pcd));

    res.normalize();

    return res;
}

extern "C" PointCloudData getGPUpcd(std::vector<Vec3> positions, std::vector<Vec3> normals, std::vector<char> materialIndex, std::vector<CudaMaterial> materialList)
{
    PointCloudData res;

    // Allouer de la mémoire sur le GPU pour les champs positions, normals, materialIndex et materialList de la structure PointCloudData
    hipMalloc(&(res.positions), positions.size() * sizeof(Vec3));
    hipMalloc(&(res.normals), normals.size() * sizeof(Vec3));
    hipMalloc(&(res.materialIndex), materialIndex.size() * sizeof(char));
    hipMalloc(&(res.materialList), materialList.size() * sizeof(CudaMaterial));

    // Copier les données depuis le CPU vers le GPU
    hipMemcpy(res.positions, (void *)positions.data(), positions.size() * sizeof(Vec3), hipMemcpyHostToDevice);
    hipMemcpy(res.normals, (void *)normals.data(), normals.size() * sizeof(Vec3), hipMemcpyHostToDevice);
    hipMemcpy(res.materialIndex, (void *)materialIndex.data(), materialIndex.size() * sizeof(char), hipMemcpyHostToDevice);
    hipMemcpy(res.materialList, (void *)materialList.data(), materialList.size() * sizeof(CudaMaterial), hipMemcpyHostToDevice);

    std::cout << "Start kd-tree building" << std::endl;
    // Construire l'arbre kd à partir des positions sur le CPU et l'envoyer sur le GPU
    std::vector<KdTreeNode> my_kd_tree = make_kd_tree(positions);
    res.kdTree = send_kd_tree(my_kd_tree);
    std::cout << "End kd-tree building" << std::endl;

    return res;
}

/*#########################################*/
/*               RayMarching              #*/
/*#########################################*/

__device__ CudaVec3 computeColor(CudaVec3 positionCamera, CudaVec3 positionPoint, CudaVec3 normalePoint, CudaVec3 positionLumiere, CudaMaterial &material)
{
    // Calcul de la direction de la lumière
    CudaVec3 directionLumiere = positionLumiere - positionPoint;
    directionLumiere.normalize();

    // Calcul de la direction de la caméra
    CudaVec3 directionCamera = positionCamera - positionPoint;
    directionCamera.normalize();

    // Calcul de la normale moyenne
    CudaVec3 normaleMoyenne = directionLumiere + directionCamera;
    normaleMoyenne.normalize();

    // Calcul de l'intensité de la lumière en utilisant l'éclairage de Blinn-Phong
    float intensiteLumiere = fmax(dot(normalePoint, directionLumiere), 0.0f);
    float specular = pow(fmax(dot(normalePoint, normaleMoyenne), 0.0f), material.SPECULAR_EXPONENT);

    // Calcul de la couleur finale en utilisant les composantes ambiante, diffuse et spéculaire
    CudaVec3 couleurFinale = material.AMBIANT_COLOR + intensiteLumiere * material.DIFFUSE_COLOR + specular * material.SPECULAR_COLOR;

    return couleurFinale;
}

__device__ CudaVec3 computeReflection(const CudaVec3 &ray, const CudaVec3 &normal)
{
    // Calculer la composante parallèle et perpendiculaire du rayon par rapport à la normale
    float dotProduct = dot(ray, normal);
    CudaVec3 parallel = normal * dotProduct;
    CudaVec3 perpendicular = ray - parallel;

    // Calculer le vecteur de réflexion
    CudaVec3 reflection = perpendicular - parallel;
    reflection.normalize();
    return reflection;
}
__device__ CudaVec3 computeTransmission(const CudaVec3 &ray, const CudaVec3 &normal, const CudaVec3 &intersection, float refractionIndex)
{
    // Calculer la composante parallèle et perpendiculaire du rayon par rapport à la normale
    float dotProduct = dot(ray, normal);
    CudaVec3 parallel = normal * dotProduct;
    CudaVec3 perpendicular = ray - parallel;

    // Calculer l'indice de réfraction du milieu d'où vient le rayon (ici, l'air avec un indice de réfraction de 1)
    float n1 = 1.0;
    // Calculer l'indice de réfraction du matériau au point d'intersection
    float n2 = refractionIndex;

    // Calculer le coefficient de réflexion et de transmission
    float reflectionCoefficient = (n1 - n2) / (n1 + n2);
    reflectionCoefficient *= reflectionCoefficient;
    // float transmissionCoefficient = 1.0 - reflectionCoefficient;

    // Calculer le vecteur de transmission
    CudaVec3 transmission = (perpendicular * n1 - normal * sqrt(1.0 - reflectionCoefficient)) * n2 + parallel;
    transmission.normalize();
    return transmission;
}
__device__ CudaVec3 mix(CudaVec3 color1, CudaVec3 color2, float mixFactor)
{
    return color1 * (1.0 - mixFactor) + color2 * mixFactor;
}

__global__ void cuda_ray_trace(int startIndex, float *rayPos, float *rayDir, float *image, int imgSize, PointCloudData pcd, int maxTransparencyIterations = 0)
{
    int index = getGlobalIdx_1D_2D()+startIndex;

    if (index < imgSize)
    {
        CudaVec3 pos = CudaVec3(rayPos[0], rayPos[1], rayPos[2]);
        CudaVec3 dir = CudaVec3(rayDir[index * 3 + 0], rayDir[index * 3 + 1], rayDir[index * 3 + 2]);

        cIntersection it = intersect(pos, dir, pcd);

        if (it.intersected)
        {
            int nearestPoint = findNearest(pcd.kdTree, it.position);

            CudaVec3 norm = normale(it.position, pcd);

            CudaVec3 color = computeColor(pos, it.position, norm, CudaVec3(0, 4, 0), pcd.materialList[pcd.materialIndex[nearestPoint]]);

            // Transparence
            float transparency = pcd.materialList[pcd.materialIndex[nearestPoint]].transparency;

            // Nombre d'itérations de transparence
            int transparencyIterations = 0;

            while (transparency > 0.0 && transparencyIterations < maxTransparencyIterations)
            {
                // Calculer le vecteur de transmission
                dir = computeTransmission(dir, norm, it.position, pcd.materialList[pcd.materialIndex[nearestPoint]].refractionIndex);
                // Lancer un nouveau rayon de transmission
                cIntersection it2 = intersect(it.position + dir * 0.01, dir, pcd);
                if (it2.intersected)
                {
                    // Si le nouveau rayon de transmission intersecte un autre objet, combiner les couleurs
                    int nearestPoint2 = findNearest(pcd.kdTree, it2.position);
                    norm = normale(it2.position, pcd);
                    CudaVec3 color2 = computeColor(it.position, it2.position, norm, CudaVec3(1, 1, 1), pcd.materialList[pcd.materialIndex[nearestPoint2]]);
                    color = mix(color, color2, transparency);
                    // Mettre à jour la transparence et le nombre d'itérations
                    transparency = pcd.materialList[pcd.materialIndex[nearestPoint2]].transparency;
                    transparencyIterations++;
                }
                else
                {
                    // Sinon, utiliser la couleur du fond
                    color = mix(color, CudaVec3(0.1, 0.1, 0.1), transparency);
                    // Mettre à jour la transparence et le nombre d'itérations
                    transparency = 0.0;
                    transparencyIterations++;
                }
            }

            image[index * 3 + 0] = color[0] > 1.0 ? 1.0 : color[0] < 0.0 ? 0.0
                                                                         : color[0];
            image[index * 3 + 1] = color[1] > 1.0 ? 1.0 : color[1] < 0.0 ? 0.0
                                                                         : color[1];
            image[index * 3 + 2] = color[2] > 1.0 ? 1.0 : color[2] < 0.0 ? 0.0
                                                                         : color[2];
        }
        else
        {
            image[index * 3 + 0] = 0.1;
            image[index * 3 + 1] = 0.1;
            image[index * 3 + 2] = 0.1;
        }
    }
}

extern "C" void cuda_ray_trace_from_camera(int w, int h, float ModelViewMatrix[16], float projectionMatrix[16], Vec3 cameraPos, PointCloudData pcd)
{

    std::vector<float> image(3 * w * h, 0.5f);
    std::vector<float> rayDir(3 * w * h);

    std::cout << "Ray tracing a " << w << " x " << h << " image" << std::endl;

    // Init
    Vec3 pos = cameraPos;

    for (int y = 0; y < h; y++)
    {
        for (int x = 0; x < w; x++)
        {
            float u = ((float)(x) + (float)(rand()) / (float)(RAND_MAX)) / w;
            float v = ((float)(y) + (float)(rand()) / (float)(RAND_MAX)) / h;

            Vec3 dir = screen_space_to_worldSpace(u, v, ModelViewMatrix, projectionMatrix) - pos;
            dir.normalize();

            rayDir[3 * (y * w + x) + 0] = dir[0];
            rayDir[3 * (y * w + x) + 1] = dir[1];
            rayDir[3 * (y * w + x) + 2] = dir[2];
        }
    }

    float *cudaDirTab;
    hipMalloc(&cudaDirTab, 3 * rayDir.size() * sizeof(float));

    float *cudaImage;
    hipMalloc(&cudaImage, 3 * image.size() * sizeof(float));

    float *hipPos;
    hipMalloc(&hipPos, 3 * sizeof(float));

    hipMemcpy(cudaDirTab, (void *)rayDir.data(), rayDir.size() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(cudaImage, (void *)image.data(), image.size() * sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(hipPos, &pos, 3 * sizeof(float), hipMemcpyHostToDevice);

    std::cout << "w : " << w << " h : " << h << " w*h : " << w * h << std::endl;

    int nbth = 5;

    // cuda_ray_trace<<<h,w>>>(hipPos, cudaDirTab, cudaImage, w);

    int nbBlock = std::ceil((w * h) / (nbth * nbth));
    // nbBlock = 5;

    std::cout << "Nb block : " << nbBlock << std::endl;

    dim3 threadsPerBlock(nbth, nbth);

    
    // dim3 numBlocks(nbBlock, 1);
    // cuda_ray_trace<<<numBlocks, threadsPerBlock>>>(hipPos, cudaDirTab, cudaImage, h * w, pcd);
    // hipMemcpy((void *)image.data(), (void *)cudaImage, image.size() * sizeof(float), hipMemcpyDeviceToHost);
    

    int nbsteps = 10;   
    int blockPerSteps = nbBlock/nbsteps;
    dim3 numBlocks(blockPerSteps, 1);

    int nextInd = 0;
    int i = 0;
    while(nextInd < nbBlock * (nbth * nbth)){
        cuda_ray_trace<<<numBlocks, threadsPerBlock>>>(nextInd, hipPos, cudaDirTab, cudaImage, h * w, pcd);
        hipDeviceSynchronize();
        i += 100/nbsteps;
        std::cout<<i<<"%"<<std::endl;
        nextInd += blockPerSteps * (nbth * nbth);
    }

    hipMemcpy((void *)image.data(), (void *)cudaImage, image.size() * sizeof(float), hipMemcpyDeviceToHost);


    std::string filename = "./rendu.ppm";

    std::ofstream f(filename.c_str(), std::ios::binary);

    if (f.fail())
    {
        std::cout << "Could not open file: " << filename << std::endl;
        return;
    }
    f << "P3" << std::endl
      << w << " " << h << std::endl
      << 255 << std::endl;
    for (int i = 0; i < w * h; i++)
        f << (int)(255.f * std::min<float>(1.f, image[i * 3 + 0])) << " " << (int)(255.f * std::min<float>(1.f, image[i * 3 + 1])) << " " << (int)(255.f * std::min<float>(1.f, image[i * 3 + 2])) << " ";
    f << std::endl;
    f.close();

    // Reset img
    image.clear();
    image.resize(w * h * 3);
    fill(image.begin(), image.end(), 0.0f);
}



/*
            // Transparence
            float transparency = pcd.materialList[pcd.materialIndex[nearestPoint]].transparency;

            // Nombre d'itérations de transparence
            int transparencyIterations = 0;

            while (transparency > 0.0 && transparencyIterations < maxTransparencyIterations)
            {
                // Calculer le vecteur de transmission
                dir = computeTransmission(dir, norm, it.position, pcd.materialList[pcd.materialIndex[nearestPoint]].refractionIndex);
                // Lancer un nouveau rayon de transmission
                cIntersection it2 = intersect(it.position + dir * 0.01, dir, pcd);
                if (it2.intersected)
                {
                    // Si le nouveau rayon de transmission intersecte un autre objet, combiner les couleurs
                    int nearestPoint2 = findNearest(pcd.kdTree, it2.position);
                    norm = normale(it2.position, pcd);
                    CudaVec3 color2 = computeColor(it.position, it2.position, norm, CudaVec3(1, 1, 1), pcd.materialList[pcd.materialIndex[nearestPoint2]]);
                    color = mix(color, color2, transparency);
                    // Mettre à jour la transparence et le nombre d'itérations
                    transparency = pcd.materialList[pcd.materialIndex[nearestPoint2]].transparency;
                    transparencyIterations++;
                }
                else
                {
                    // Sinon, utiliser la couleur du fond
                    color = mix(color, CudaVec3(0.1, 0.1, 0.1), transparency);
                    // Mettre à jour la transparence et le nombre d'itérations
                    transparency = 0.0;
                    transparencyIterations++;
                }
            }
            */