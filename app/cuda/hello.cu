#include "hip/hip_runtime.h"
#include <hello.cuh>

__global__ void cuda_hello()
{
	printf("Hello World du GPU\n");
}

void kernel()
{
	printf("Hello World du CPU\n");

	// Launch CUDA kernel.
	cuda_hello<<<1, 1>>>();

	hipDeviceSynchronize();
}