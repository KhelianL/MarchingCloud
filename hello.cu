#include <stdio.h>
#include <stdlib.h>

#include <hip/hip_runtime.h>


__global__ void cuda_hello(){
	printf("Hello World du GPU\n");
}


int main2() {
	printf("Hello World du CPU\n");
	cuda_hello<<<1,1>>>();
    hipDeviceSynchronize();

	return EXIT_SUCCESS;
}